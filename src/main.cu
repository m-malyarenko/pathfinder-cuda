﻿#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <errno.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.cuh"
#include "pathfinder.cuh"
#include "generator.cuh"

unsigned int BLOCK_DIM = 0;
unsigned int GRID_DIM = 0;

__host__ int main(int argc, const char* argv[]) {
	/*----------------------------------------------------------------------------
	 * Parameter Check & Inial Setup
	 *--------------------------------------------------------------------------*/

	/* Check Input Parameters */
	if (argc != 4) {
		fprintf(stderr, "ERROR: incorrect numer of arguments\n");
		return EXIT_FAILURE;
	}

	long field_size_l = strtol(argv[1], NULL, 10);
	if ((errno == ERANGE) ||
		(field_size_l < MIN_FIELD_SIZE) ||
		(field_size_l > MAX_FIELD_SIZE))
	{
		fprintf(stderr, "ERROR: incorrect field size parameter\n");
		return EXIT_FAILURE;
	}

	GRID_DIM = strtol(argv[3], NULL, 10);
	if ((errno == ERANGE) ||
		(BLOCK_DIM < MIN_BLOCK_DIM) ||
		(BLOCK_DIM > MAX_BLOCK_DIM))
	{
		fprintf(stderr, "ERROR: incorrect block dim parameter\n");
		return EXIT_FAILURE;
	}

	BLOCK_DIM = strtol(argv[3], NULL, 10);
	if ((errno == ERANGE) ||
		(BLOCK_DIM < MIN_BLOCK_DIM) ||
		(BLOCK_DIM > MAX_BLOCK_DIM))
	{
		fprintf(stderr, "ERROR: incorrect block dim parameter\n");
		return EXIT_FAILURE;
	}

	/* Check Device Properties */
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ERROR: hipSetDevice failed\n");
		return EXIT_FAILURE;
	}

	hipDeviceProp_t property = {0};
	hipGetDeviceProperties(&property, 0);
	size_t device_memory = property.totalGlobalMem;
	if (device_memory < (MAX_FIELD_SIZE * MAX_FIELD_SIZE)) {
		fprintf(stderr, "ERROR: not enough global memory\n");
		return EXIT_FAILURE;
	}

	/* Seed Rand */
	srand(time(0));

	/*----------------------------------------------------------------------------
	 * Variables & Constants Declaration
	 *--------------------------------------------------------------------------*/

	/* Common Constants */
	const size_t field_size = (size_t) field_size_l;
	const size_t field_bytes = (size_t) field_size * (size_t) field_size * sizeof(unsigned int);

	#ifndef TEST_MODE
	const size_t contour_num = (size_t) (1 + ((size_t) rand() % MAX_CONTOUR_NUM));
	#else
	const size_t contour_num = 3;
	#endif

	/* Global Host Variables */
	int* h_field = NULL;
	contour_instance* h_contour_list = NULL;
	point2d h_start = { 0 };
	point2d h_finish = { 0 };

	/* Device Variables */
	int* d_field_a = NULL;
	int* d_field_b = NULL;
	contour_instance* d_contour_list = NULL;
	point2d* d_start = NULL;
	point2d* d_finish = NULL;

	/* Generator Variables */
	generator_param gen_parameters;
	memset(&gen_parameters, 0, sizeof(generator_param));

	/* Pathfinder Variables */
	pathfinder_param pathfinder_parameters;
	memset(&pathfinder_parameters, 0, sizeof(pathfinder_param));
	bool path_exists = false;

	/* Timer Instances */
	hipEvent_t timer_start;
	hipEventCreate(&timer_start);
	hipEvent_t timer_stop;
	hipEventCreate(&timer_stop);
	float generator_time = 0.F;
	float pathfinder_time = 0.F;
	float backtrace_time = 0.F;

	/*----------------------------------------------------------------------------
	 * Memory Management
	 *--------------------------------------------------------------------------*/

	/* Host Memory */
	cudaStatus = hipHostAlloc(&h_field, field_bytes, hipHostMallocDefault);
	HANDLE_ERROR(cudaStatus, "Failed to allocate field on the host\n");

	cudaStatus = hipHostAlloc(&h_contour_list, contour_num * sizeof(contour_instance), hipHostMallocDefault);
	HANDLE_ERROR(cudaStatus, "Failed to allocate contour list\n");

	/* Device Memory */
	cudaStatus = hipMalloc(&d_field_a, field_bytes);
	HANDLE_ERROR(cudaStatus, "Failed to allocate field on the device\n");
	cudaStatus = hipMemset(d_field_a, 0, field_bytes);
	HANDLE_ERROR(cudaStatus, "Memset failed\n");

	cudaStatus = hipMalloc(&d_field_b, field_bytes);
	HANDLE_ERROR(cudaStatus, "Failed to allocate field on the device\n");

	cudaStatus = hipMalloc(&d_contour_list, contour_num * sizeof(contour_instance));
	HANDLE_ERROR(cudaStatus, "Failed to allocate contour list on the device\n");

	cudaStatus = hipMalloc(&d_start, sizeof(point2d));
	HANDLE_ERROR(cudaStatus, "Failed to allocate start point on the device\n");

	cudaStatus = hipMalloc(&d_finish, sizeof(point2d));
	HANDLE_ERROR(cudaStatus, "Failed to allocate finish point on the device\n");

	/*----------------------------------------------------------------------------
	 * Generate Barrier Contours
	 *--------------------------------------------------------------------------*/

	printf("----------------[LOG]----------------\n\n");
	printf("Pathfinder: Start...\n");
	printf("Pathfinder: Generating field...\n");

	h_contour_list[0].x = field_size / 2;
	h_contour_list[0].y = 0;
	h_contour_list[0].size = field_size / 2;

	h_contour_list[1].x = 0;
	h_contour_list[1].y = field_size / 4;
	h_contour_list[1].size = field_size / 4;

	h_contour_list[2].x = field_size / 4;
	h_contour_list[2].y = 3 * (field_size / 4);
	h_contour_list[2].size = field_size / 4;

	// for (size_t i = 0; i < contour_num; i++) {
	// 	generator_new_contour(field_size, &(h_contour_list[i]));
	// }

	cudaStatus = hipMemcpy(d_contour_list,
		                    h_contour_list,
		                    contour_num * sizeof(contour_instance),
		                    hipMemcpyDefault);
	HANDLE_ERROR(cudaStatus, "Failed to copy data Host -> Device\n");

	gen_parameters.d_field = d_field_a;
	gen_parameters.field_size = field_size;
	gen_parameters.d_contour_list = d_contour_list;
	gen_parameters.contour_list_size = contour_num;

	/* Start Timer */
	hipEventRecord(timer_start, 0);

	/* Run Generator */
	cudaStatus = generator_exec(&gen_parameters);
	HANDLE_ERROR(cudaStatus, "Generator failed\n");

	/* Stop Timer */
	hipEventRecord(timer_stop, 0);
	hipEventSynchronize(timer_stop);
	hipEventElapsedTime(&generator_time, timer_start, timer_stop);

	cudaStatus = hipMemcpy(h_field, d_field_a, field_bytes, hipMemcpyDefault);
	HANDLE_ERROR(cudaStatus, "Failed to copy data Device -> Host\n");

	printf("Pathfinder: Field generated\n");

	/*----------------------------------------------------------------------------
	 * Generate Start & Finish points
	 *--------------------------------------------------------------------------*/

	printf("Pathfinder: Generating start/finish points...\n");

	memset(&h_start, 0, sizeof(point2d));
	memset(&h_finish, 0, sizeof(point2d));

GEN_START_FINISH:

	h_start.row = 1;
	h_start.col = 1;

	h_finish.row = field_size - 1;
	h_finish.col = 0;

	// h_start.row = rand() % field_size;
	// h_start.col = rand() % field_size;

	// h_finish.row = rand() % field_size;
	// h_finish.col = rand() % field_size;

	if ((h_start.row == h_finish.row) && (h_start.col == h_finish.col)) {
		goto GEN_START_FINISH;
	}

	/* Check if Target Points are inside the Contour */
	if (!pathfinder_check_target_points(d_field_a, field_size, &h_start, &h_finish)) {
		goto GEN_START_FINISH;
	}

	cudaStatus = hipMemcpy(d_start, &h_start, sizeof(point2d), hipMemcpyDefault);
	HANDLE_ERROR(cudaStatus, "Failed to copy data Host -> Device\n");

	cudaStatus = hipMemcpy(d_finish, &h_finish, sizeof(point2d), hipMemcpyDefault);
	HANDLE_ERROR(cudaStatus, "Failed to copy data Host -> Device\n");

	printf("Pathfinder: Start/finish points generated\n");

	/*----------------------------------------------------------------------------
	 * Find Path on the Field
	 *--------------------------------------------------------------------------*/

	printf("Pathfinder: Scanning the field...\n");

	cudaStatus = pathfinder_set_start_val(d_field_a, field_size, &h_start);
	HANDLE_ERROR(cudaStatus, "Failed to set start value\n");

	pathfinder_parameters.d_field_a = d_field_a;
	pathfinder_parameters.d_field_b = d_field_b;
	pathfinder_parameters.field_size = field_size;
	pathfinder_parameters.d_start = d_start;
	pathfinder_parameters.d_finish = d_finish;

	/* Start Timer */
	hipEventRecord(timer_start, 0);

	/* Run Pathfinder*/
	path_exists = pathfinder_exec(&pathfinder_parameters);

	/* Stop Timer */
	hipEventRecord(timer_stop, 0);
	hipEventSynchronize(timer_stop);
	hipEventElapsedTime(&pathfinder_time, timer_start, timer_stop);

	printf("Pathfinder: Field is scanned\n");

	if (path_exists) {
		printf("Pathfinder: Backtracing the path...\n");

		cudaStatus = hipMemcpy(h_field,
			                    d_field_b,
			                    field_bytes,
			                    hipMemcpyDefault);
		HANDLE_ERROR(cudaStatus, "Failed to copy data Device -> Host\n");

		/* Start Timer */
		hipEventRecord(timer_start, 0);

		/* Run Path Backtrace */
		pathfinder_backtrace(h_field, field_size, &h_start, &h_finish);

		/* Stop Timer */
		hipEventRecord(timer_stop, 0);
		hipEventSynchronize(timer_stop);
		hipEventElapsedTime(&backtrace_time, timer_start, timer_stop);

		printf("Pathfinder: Path backtracing done\n");
	}
	else {
		fprintf(stderr, "No path avaliable\n");
		goto ERROR;
	}

	printf("Pathfinder: Finish\n\n");

	/*----------------------------------------------------------------------------
	 * Print Statistics
	 *--------------------------------------------------------------------------*/
	printf("---------------[STATS]---------------\n\n");
	printf("Field generation time: %.3f milliseconds\n", generator_time);
	printf("Path finding time: %.3f milliseconds\n", pathfinder_time);
	printf("Path backtrace time: %.3f milliseconds\n", backtrace_time);
	printf("Total time: %.3f milliseconds\n", generator_time + pathfinder_time + backtrace_time);

#ifndef NDEBUG
	/* Print Out Field */
	printf("+");
	for (unsigned int i = 0; i < field_size * 2; i++) {
		printf("-");
	}
	printf("+\n");

	for (unsigned int i = 0; i < field_size; i++) {
		printf("|");
		for (unsigned int j = 0; j < field_size; j++) {
			if ((i == h_start.row) && (j == h_start.col)) {
				printf("S ");
			}
			else if ((i == h_finish.row) && (j == h_finish.col)) {
				printf("F ");
			}
			else {
				int point_val = h_field[field_size * i + j];
				if (point_val == BARRIER_VAL) {
					printf("# ");
				}
				else if (point_val == TRACE_VAL) {
					printf("+ ");
				}
				else {
					printf("  ");
				}
			}
		}
		printf("|\n");
	}

	printf("+");
	for (unsigned int i = 0; i < field_size * 2; i++) {
		printf("-");
	}
	printf("+\n");
#endif /* !NDEBUG */

ERROR:

	hipFree(d_field_a);
	hipFree(d_field_b);
	hipFree(d_contour_list);
	hipFree(d_start);
	hipFree(d_finish);
	hipHostFree(h_field);
	hipHostFree(h_contour_list);
}
